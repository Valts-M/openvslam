#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace openvslam { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
